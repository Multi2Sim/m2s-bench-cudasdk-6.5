#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "repeat.h"
using namespace std;



__global__ void int_add (int *c, int *a, int *b)
{

  int i = *a;
  int j = *b;
  int k0, k1, k2, k3, k4, k5, k6;
  //k6, k7, k8, k9, k10, k11, k12, k13, k14, k15, k16,
  //k17, k18, k19, k20, k21, k22, k23, k24, k25, k26, k27, k28, k29, k30;
/*
  asm volatile (
 		"add.s32 %0, %1, %2;\n\t"
 	  : "=r"(k0) : "r"(i) , "r"(j)
 	  );

   asm volatile (
 		"add.s32 %0, %1, %2;\n\t"
 	  : "=r"(k1) : "r"(i) , "r"(j)
 	  );

   asm volatile (
 		"add.s32 %0, %1, %2;\n\t"
 	  : "=r"(k2) : "r"(i) , "r"(j)
 	  );

   asm volatile (
 		"add.s32 %0, %1, %2;\n\t"
 	  : "=r"(k3) : "r"(i) , "r"(j)
 	  );

   asm volatile (
 		"add.s32 %0, %1, %2;\n\t"
 	  : "=r"(k4) : "r"(i) , "r"(j)
 	  );


   asm volatile (
  		"sub.s32 %0, %1, %2;\n\t"
  	  : "=r"(k5) : "r"(i) , "r"(j)
  	  );
*/
  repeat25(
  asm volatile (
		"add.s32 %0, %0, %1;\n\t"
	  : "=r"(k0) : "r"(i) , "r"(j)
	  );

  asm volatile (
		"add.s32 %0, %0, %1;\n\t"
	  : "=r"(k1) : "r"(i) , "r"(j)
	  );

  asm volatile (
		"add.s32 %0, %0, %1;\n\t"
	  : "=r"(k2) : "r"(i) , "r"(j)
	  );

  asm volatile (
		"add.s32 %0, %0, %1;\n\t"
	  : "=r"(k3) : "r"(i) , "r"(j)
	  );

  asm volatile (
		"add.s32 %0, %0, %1;\n\t"
	  : "=r"(k4) : "r"(i) , "r"(j)
	  );


  asm volatile (
 		"sub.s32 %0, %1, %0;\n\t"
 	  : "=r"(k5) : "r"(i) , "r"(j)
 	  );
  )
 // printf("k = %d\n", k);
//  *c= k0;
 // *c = k1;
 // *c = k2;
 // *c = k3;
  //*c = k4;

  k6 = k5;
  k5 = k4;
  k4 = k3;
  k3 = k2;
  k2= k1;
  k1= k0;
  *c = k1;
}

int main()
{
	int length = sizeof(int);
	int *a = (int*)malloc(length);
	int *b = (int*)malloc(length);
	int *c = (int*)malloc(length);

	*a = 1;
	*b = 1;
	int *ad, *bd, *cd;
	// initialize device memory
	hipMalloc( (void**)&ad, length);
	hipMalloc( (void**)&bd, length);
	hipMalloc( (void**)&cd, length);

	// copy data to device
	hipMemcpy( ad, a, sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( bd, b, sizeof(int), hipMemcpyHostToDevice );

	// setup block and grid size	
	int_add<<<2000, 256>>>(cd, ad, bd);
	hipMemcpy( c, cd, sizeof(int), hipMemcpyDeviceToHost );

	cout << "Kernel Finished" << endl;
	//Verify
/*
	bool flag;
	if (*c == 21)
		flag = true;
	else
	{
		flag = false;
		printf("result is %d\n", *c);
	}
	if(flag)
		cout << "Verification passes." << endl;
	else
		cout << "Verification fails." << endl;

*/
	// free device memory
	hipFree( ad );
	hipFree( bd );
	hipFree (cd);
	return EXIT_SUCCESS;
}

