
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#define NThreads 64
#define NBlocks 16

#define Num NThreads*NBlocks

__global__ void bitonic_sort_step(int *arr, int i, int j)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tid_comp = tid ^ j;
	if (tid_comp > tid)
	{
		if ((tid & i) == 0)
		{
			//ascending
			if (arr[tid] > arr[tid_comp])
			{
				int temp = arr[tid];
				arr[tid] = arr[tid_comp];
				arr[tid_comp] = temp;
			}
		}
		else
		{
			//descending
			if (arr[tid] < arr[tid_comp])
			{
				int temp = arr[tid];
				arr[tid] = arr[tid_comp];
				arr[tid_comp] = temp;
			}

		}
	}
}

int main(int argc, char* argv[])
{
    int* arr= (int*) malloc(Num*sizeof(int));
    int* arr_temp = (int*) malloc(Num*sizeof(int));

    // Initialization
    time_t t;
    srand((unsigned)time(&t));
    for(int i=0;i<Num;i++){
        arr[i] = rand() % 10000;
    	//arr[i] = i;
     }

    //init device variable
    int* dev_ptr;
    hipMalloc((void**)&dev_ptr,Num*sizeof(int));
    hipMemcpy(dev_ptr,arr,Num*sizeof(int),hipMemcpyHostToDevice);

 /*
    for(int i=0;i<Num;i++)
    {
        printf("%d\t",arr[i]);
    }
 */
    printf("\n End initialization \n");


    dim3 blocks(NBlocks,1);
    dim3 threads(NThreads,1);

    // bitonic sort
    for(unsigned int i=2; i<=Num; i<<=1)
    {
    	// bitonic merge
        for(unsigned int j=i>>1; j>0; j>>=1)
        {
        	bitonic_sort_step<<<blocks,threads>>>(dev_ptr,i,j);
   /*     	cudaMemcpy(arr_temp,dev_ptr,Num*sizeof(int),cudaMemcpyDeviceToHost);
            for(int i=0;i<Num;i++){
                printf("%d\t",arr_temp[i]);
            }
            printf("\n");
    */
        }
    }

    hipMemcpy(arr,dev_ptr,Num*sizeof(int),hipMemcpyDeviceToHost);

    // Self validation
    bool flag = true;
    for(int i = 0;i < Num - 1;i++)
    {
    	if (arr[i] > arr[i+1])
    	{
    		flag = false;
    		break;
    	}
    }

    if (flag)
    	printf("\nVerification passes\n");
    else
    	printf("\nVerification fails\n");

    hipFree(dev_ptr);
    return 0;
}
