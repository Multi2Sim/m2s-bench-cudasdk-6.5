#include "hip/hip_runtime.h"
#include"stdio.h"
//#include "matrixmul.cuh"

#define BLOCK_SIZE 	16
#define A_HEIGHT    	128
#define A_WIDTH		128
#define B_HEIGHT	128
#define B_WIDTH		128
#define C_HEIGHT	A_HEIGHT
#define C_WIDTH		B_WIDTH

__global__ void matrix_mulKernel(int *c, int *a, int *b,
		int a_height,int a_width, int b_width, int c_width)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if(row >= a_height || col >= b_width) return;
	int i, sum=0;
	for(i =0; i<a_width; ++i)
	{
		sum += a[row * a_width + i] * b[i * b_width + col];
	}
	c[row * c_width + col] =sum;
}

void matrix_multiplication(const int *a, const int *b, int *c, int a_hiehgt, int b_width, int b_height);

void print_matrix( int *matrix, int height, int width);

int main()
{
	int *a = (int*) calloc(A_HEIGHT * A_WIDTH, sizeof(unsigned int) );
	int *b = (int*) calloc(B_HEIGHT * B_WIDTH, sizeof(unsigned int) );
	int *c = (int*) calloc(C_HEIGHT * C_WIDTH, sizeof(unsigned int) );
	int *d = (int*) calloc(C_HEIGHT * C_WIDTH, sizeof(unsigned int) );
	int i;
	for (i =0; i < A_HEIGHT * A_WIDTH; i++)
	{
		a[i] = i;
		b[i] = i;
	}

    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    hipMalloc((void**)&dev_c, C_HEIGHT * C_WIDTH * sizeof(unsigned int));


    hipMalloc((void**)&dev_a, A_HEIGHT * A_WIDTH * sizeof(unsigned int));


    hipMalloc((void**)&dev_b, B_HEIGHT * B_WIDTH * sizeof(unsigned int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, A_HEIGHT * A_WIDTH * sizeof(unsigned int), hipMemcpyHostToDevice);

    hipMemcpy(dev_b, b, B_HEIGHT * B_WIDTH * sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 dimBlock (BLOCK_SIZE,BLOCK_SIZE); // block( blockIdx, blockIDy)
    dim3 grid ((B_WIDTH + dimBlock.x - 1) / dimBlock.x,
    		(A_HEIGHT + dimBlock.y - 1) / dimBlock.y);		// grid(gloalsizeX + blockidx -1)/blockidx,gloalsizeY + blockidy -1)/blockidy);

    matrix_mulKernel<<<grid, dimBlock>>>(dev_c, dev_a, dev_b, A_HEIGHT, A_WIDTH, B_WIDTH, C_WIDTH);

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, C_HEIGHT * C_WIDTH * sizeof(int), hipMemcpyDeviceToHost);


    matrix_multiplication(a, b ,d, A_HEIGHT, B_WIDTH, B_HEIGHT);

    bool flag = true;
    for(int i = 0; i < A_HEIGHT * B_WIDTH; i++ )
{

	if (c[i] != d[i])
	{
		printf("Verification fail\n");
		flag = false;
		break;
	}

}
    if (flag)
    	printf("Verification pass\n");
	//printf("Matrix A:\n");
	//print_matrix(a, size);
	//printf("Matrix B:\n");
	//print_matrix(b, size);
	printf("Matrix C:\n");
	print_matrix(c, 10, 10);
	printf("Matrix D:\n");
    	print_matrix(d,10, 10);
}


void matrix_multiplication(const int *a, const int *b, int *c, int a_height, int b_width, int b_height)
{
	for(int i = 0; i < a_height; i++)
	{
		for(int j = 0; j < b_width; j++)
		{
			int sum = 0;
			for(int k = 0; k < b_height ; k++)
			{
				sum+= a[i * b_height + k] * b[k * b_width + j];

			}
			c[i * b_width + j] = sum;
		}
	}
}

void print_matrix( int *matrix, int height, int width)
{
	int i , j;
	for(i = 0; i < width; i++)
{
	for(j = 0; j < height; j++)
		printf("%5d", matrix[i * width + j]);
	printf("\n");
}
}
